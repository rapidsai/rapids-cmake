/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2023, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

int main()
{
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  // We should have anywhere from 1 to 2 devices allocated
  std::cout << "Seeing " << nDevices << " GPU devices" << std::endl;

  if (nDevices == 0 || nDevices > 2) { return 1; }
  return 0;
}
