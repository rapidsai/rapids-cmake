/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2023, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

int main()
{
  // Very we only have a single GPU visible to us
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  if (nDevices == 0) { return 1; }
  std::cout << "Seeing at least a single GPU" << std::endl;
  return 0;
}
