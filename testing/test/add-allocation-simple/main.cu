/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2023, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

int main()
{
  // Verify we only have a single GPU visible to us
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);

  std::cout << "Seeing " << nDevices << " GPU devices" << std::endl;

  if (nDevices == 0 || nDevices > 3) { return 1; }
  return 0;
}
