
#include <hip/hip_runtime.h>
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2024, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

static __global__ void example_cuda_kernel(int& r, int x, int y) { r = x * y + (x * 4 - (y / 2)); }

int static_launch_kernelC(int x, int y)
{
  int r;
  example_cuda_kernel<<<1, 1>>>(r, x, y);
  return r;
}
