/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2023, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */


#include <hip/hip_runtime.h>
#include <iostream>

int static_launch_kernelA(int x, int y);
int static_launch_kernelB(int x, int y);

int main(int argc, char**)
{
  auto resultA = static_launch_kernelA(3, argc);
  auto resultB = static_launch_kernelB(3, argc);
  if (resultA != 6 && resultB != 6) { return 1; }
  return 0;
}
